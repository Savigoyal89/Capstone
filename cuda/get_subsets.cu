#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
#define THREADS_PER_BLOCK 1024
#define MAX_ELEMENT 7
#define NUM_ELEMENTS_SUBSET 3

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


__host__ __device__ void get_nums_array(int* arr){
    for (int i =0; i < MAX_ELEMENT;i++){
      arr[i]=i;
    }
}

__host__ __device__ int get_r_steps_product(int n, int steps) {
  int out = 1;
  for (int i = 0; i < steps; i++) {
    out = out * (n - i);
  }
  return out;
}

__host__ __device__ int nCr(int n, int r) {
  if (r == 0) {
    return 1;
  } else if (r == 1) {
    return n;
  } else {
    return (get_r_steps_product(n, r) / get_r_steps_product(r, r));
  }
}

__host__ __device__ int convert_subset_to_number(int *subset, int subset_size) {
  int sum = 0;
  for (int i = subset_size - 1; i >= 0; i--) {
    sum = sum * MAX_ELEMENT + subset[i];
  }
  return sum;
}

__host__ __device__ void convert_number_to_subset(int number, int *subset) {
  int index = 0;
  while (number > 0) {
    subset[index] = number % MAX_ELEMENT;
    number = number / MAX_ELEMENT;
    index++;
  }
}

__host__ __device__ void get_combinations(int *arr, int arr_index, int *current, int current_index,
                      int elements_size, int subset_size, int *output,
                      int &output_index) {
  // Current combination is ready, print it
  if (current_index == subset_size) {
    // Copy current to output at output_index
    output[output_index] = convert_subset_to_number(current, subset_size);
    output_index++;
    return;
  }
  // When no more elements are there to put in data[]
  if (arr_index >= elements_size) return;
  // Case 1: Exclude the arr_index
  get_combinations(arr, arr_index + 1, current, current_index, elements_size,
                   subset_size, output, output_index);

  // Case 2: Include the arr_index into the current_index
  current[current_index] = arr[arr_index];
  get_combinations(arr, arr_index + 1, current, current_index + 1,
                   elements_size, subset_size, output, output_index);
}


__global__ void get_ideal_pte_combinations(int* input, int* output){
  int index = threadIdx.x ;
  int input_subset_number = input[index];
  output[index]  = input_subset_number;
}

__host__ void print_subsets(int* subsets, int num_subsets, int subset_size ){
  int subset[subset_size] = {0};
  for (int i = 0; i < num_subsets; i++) {
    convert_number_to_subset(subsets[i], subset);
    for (int j = 0; j < subset_size; j++) {
      printf("%d ", subset[j]);
    }
    printf("\n");
  }
}

__host__ void get_input_subsets(int subset_size, int* subsets){
  int subset[subset_size] = {0};
  // Calculate the output size
  int output_index = 0;
  int arr[MAX_ELEMENT] = {0};
  get_nums_array(arr);
  printf("Subset size: %d\n", subset_size);
  get_combinations(arr, 0, subset, 0, MAX_ELEMENT, subset_size, subsets,output_index);
}

int main(int argc, char *argv[]) {
  // Declare host copies
  int num_subsets = nCr(MAX_ELEMENT, NUM_ELEMENTS_SUBSET);
  int h_subsets[num_subsets] = {0};
  get_input_subsets(NUM_ELEMENTS_SUBSET, h_subsets);
  printf("Printing output from CPU with size: %d\n", num_subsets);
  print_subsets(h_subsets,num_subsets,NUM_ELEMENTS_SUBSET);
  // Initiate device copies
  int *d_subsets, *d_output;  // device copies
  int size = num_subsets * sizeof(int);

  // Allocate space for device copies
  hipMalloc((void**) &d_subsets, size);
  hipMalloc((void**) &d_output, size);

  // Copy from host to device
  hipMemcpy(d_subsets, h_subsets,size, hipMemcpyHostToDevice);

  // Run the functions on device
  get_ideal_pte_combinations<<<1,num_subsets>>>(d_subsets,d_output);

  // Copy from device to host
  h_subsets[num_subsets] = {0};
  hipMemcpy(h_subsets, d_output, size, hipMemcpyDeviceToHost);
  printf("Printing output from GPU with size: %d\n",num_subsets);
  print_subsets(h_subsets, num_subsets,NUM_ELEMENTS_SUBSET);

  // Clean up
  hipFree(d_subsets);
  hipFree(d_output);
  return 0;

}

