#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
using namespace std;
#define THREADS_PER_BLOCK 1024
#define MAX_ELEMENT 7
#define NUM_ELEMENTS_SUBSET 3
#define NUM_ELEMENTS_PARTNER 10
#define OFFSET 2

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


__host__ __device__ void get_nums_array(int* arr){
    for (int i =0; i < MAX_ELEMENT;i++){
      arr[i]=i;
    }
}

__host__ __device__ int get_r_steps_product(int n, int steps) {
  int out = 1;
  for (int i = 0; i < steps; i++) {
    out = out * (n - i);
  }
  return out;
}

__host__ __device__ int nCr(int n, int r) {
  if (r == 0) {
    return 1;
  } else if (r == 1) {
    return n;
  } else {
    return (get_r_steps_product(n, r) / get_r_steps_product(r, r));
  }
}

__host__ __device__ int convert_subset_to_number(int *subset, int subset_size) {
  int sum = 0;
  for (int i = subset_size - 1; i >= 0; i--) {
    sum = sum * MAX_ELEMENT + subset[i];
  }
  return sum;
}

__host__ __device__ void convert_number_to_subset(int number, int *subset) {
  int index = 0;
  while (number > 0) {
    subset[index] = number % MAX_ELEMENT;
    number = number / MAX_ELEMENT;
    index++;
  }
}

__host__ __device__ void get_combinations(int *arr, int arr_index, int *current, int current_index,
                      int elements_size, int subset_size, int *output,
                      int &output_index) {
  // Current combination is ready, print it
  if (current_index == subset_size) {
    // Copy current to output at output_index
    output[output_index] = convert_subset_to_number(current, subset_size);
    output_index++;
    return;
  }
  // When no more elements are there to put in data[]
  if (arr_index >= elements_size) return;
  // Case 1: Exclude the arr_index
  get_combinations(arr, arr_index + 1, current, current_index, elements_size,
                   subset_size, output, output_index);

  // Case 2: Include the arr_index into the current_index
  current[current_index] = arr[arr_index];
  get_combinations(arr, arr_index + 1, current, current_index + 1,
                   elements_size, subset_size, output, output_index);
}

__device__ int get_set_diff(int *input_elements, int* subset,int *out) {
    int i = 0;
    int j = 0;
    int k = 0;
    while (i < MAX_ELEMENT) {
        if (j<NUM_ELEMENTS_SUBSET && (input_elements[i] == subset[j])) {
            j++;
        }
        else {
            if (input_elements[i]>subset[0]) {
                out[k] = input_elements[i];
                k++;
            }
        }
        i++;
    }
  return k;
}

__device__ double get_sum_of_power(int* set, int set_size, int power) {
    double sum = 0;
    for (int i = 0; i < set_size; i++) {
        sum += pow(set[i], power);
    }
    return sum;
}

__device__ bool is_ideal_PTE(int setNum1, int setNum2) {
    if (setNum1 ==0 || setNum2 ==0){
       return false;
    }
    int set1[NUM_ELEMENTS_SUBSET] = {0};
    int set2[NUM_ELEMENTS_SUBSET] = {0};
    convert_number_to_subset(setNum1,set1);
    convert_number_to_subset(setNum2,set2);
    for (int i = 1; i < NUM_ELEMENTS_SUBSET; i++) {
        if (get_sum_of_power(set1, NUM_ELEMENTS_SUBSET, i) !=
            get_sum_of_power(set2, NUM_ELEMENTS_SUBSET, i)) {
            return false;
        }
    }
    return true;
}
__global__ void get_ideal_pte_combinations(int* input, int* output){
  int index = threadIdx.x ;
  int input_subset_number = input[index];
  int subset[NUM_ELEMENTS_SUBSET] = {0};
  convert_number_to_subset(input_subset_number, subset);
  int set_diff[MAX_ELEMENT -  NUM_ELEMENTS_SUBSET] = {0};
  int arr[MAX_ELEMENT] = {0};
  get_nums_array(arr);
  int set_diff_size = get_set_diff(arr,subset,set_diff);
  int partner_subsets[NUM_ELEMENTS_PARTNER] = {0};
  int partner_subset_index = 0;
  int current[NUM_ELEMENTS_SUBSET] = {0};
  int offset = 0;
  get_combinations(set_diff, 0, current, 0, set_diff_size, NUM_ELEMENTS_SUBSET, partner_subsets,partner_subset_index);
  for (int i=0; i < partner_subset_index;i++){
    if (is_ideal_PTE(input_subset_number, partner_subsets[i])){
        output[index + offset] = input_subset_number;
        output[index + offset + 1] = partner_subsets[i];
	offset +=2;
    }
  }
  delete[] partner_subsets;
  /*for (int i =0; i < OFFSET;i=i+2){
     output[index*OFFSET + i] = input_subset_number;
     output[index*OFFSET + i+1] = input_subset_number;
  }*/
}

__host__ void print_subsets(int* subsets, int num_subsets, int subset_size ){
  int subset[subset_size] = {0};
  for (int i = 0; i < num_subsets; i++) {
    convert_number_to_subset(subsets[i], subset);
    for (int j = 0; j < subset_size; j++) {
      printf("%d ", subset[j]);
    }
    printf("\n");
  }
}

__host__ void print_output(int* subsets, int num_subsets ){
  int subset[NUM_ELEMENTS_SUBSET] = {0};
  for (int i = 0; i < num_subsets-1; i=i+2) {
    if (subsets[i]==0 || subsets[i+1]==0){
    	continue;
    }	    
    convert_number_to_subset(subsets[i], subset);
    for (int j = 0; j < NUM_ELEMENTS_SUBSET; j++) {
      printf("%d ", subset[j]);
    }
    printf(" -- ");
    convert_number_to_subset(subsets[i+1], subset);
    for (int j = 0; j < NUM_ELEMENTS_SUBSET; j++) {
      printf("%d ", subset[j]);
    }
    printf("\n");
  }
}

__host__ void get_input_subsets(int subset_size, int* subsets){
  int subset[subset_size] = {0};
  // Calculate the output size
  int output_index = 0;
  int arr[MAX_ELEMENT] = {0};
  get_nums_array(arr);
  get_combinations(arr, 0, subset, 0, MAX_ELEMENT, subset_size, subsets,output_index);
}

int main(int argc, char *argv[]) {
  // Declare host copies
  int num_subsets = nCr(MAX_ELEMENT, NUM_ELEMENTS_SUBSET);
  int h_subsets[num_subsets] = {0};
  get_input_subsets(NUM_ELEMENTS_SUBSET, h_subsets);
  printf("Printing output from CPU with size: %d\n", num_subsets);
  print_subsets(h_subsets,num_subsets,NUM_ELEMENTS_SUBSET);
  
  // Initiate device copies
  int *d_subsets, *d_output;  // device copies
  int size = num_subsets * sizeof(int);

  // Allocate space for device copies
  hipMalloc((void**) &d_subsets, size);
  hipMalloc((void**) &d_output, size*OFFSET);

  // Copy from host to device
  hipMemcpy(d_subsets, h_subsets,size, hipMemcpyHostToDevice);

  // Run the functions on device
  get_ideal_pte_combinations<<<1,num_subsets>>>(d_subsets,d_output);

  // Copy from device to host
  int h_output[num_subsets*OFFSET] = {0};
  hipMemcpy(h_output, d_output, size*OFFSET, hipMemcpyDeviceToHost);
  printf("Printing ideal PTE output for %d element subset \n", NUM_ELEMENTS_SUBSET);
  print_output(h_output,num_subsets*OFFSET);

  // Clean up
  hipFree(d_subsets);
  hipFree(d_output);
  return 0;

}

